#include "hip/hip_runtime.h"
#include "Detect.h"
#include "Transform.h"
#include "Update_context.h"

__global__ void process(unsigned char* srcData, const float* tgtData, const int h, const int w)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = ix + iy * w;
    int idx3 = idx * 3;

    if (ix < w && iy < h)
    {
        srcData[idx3 + 0] *=  tgtData[idx];
        srcData[idx3 + 1] *= tgtData[idx];
        srcData[idx3 + 2] *=  tgtData[idx];
    }
}

RVM::RVM(char* sModelPath) //: session(env, sModelPath, session_options) 
{
	// ���CUDAExecutionProvider���Ựѡ����
	Ort::ThrowOnError(OrtSessionOptionsAppendExecutionProvider_CUDA(session_options, 0));   //CUDA���ٿ���
	session_options.SetGraphOptimizationLevel(ORT_ENABLE_BASIC); //����ͼ�Ż�����
	session_options.SetIntraOpNumThreads(nThreadNum); // �����߳�
	session = new Ort::Session(env, sModelPath, session_options);    // �����Ự����ģ�ͼ��ص��ڴ�
	Ort::AllocatorWithDefaultOptions allocator;
	
}


void RVM::detect(unsigned char* aImg,unsigned char* aResultImg, int nWeigh, int nHeight)
{
	// ������ʱ����dsr
	dynamic_dsr_value_handler.at(0) = downsample_ratio;

	// ͼ��Ԥ������������src, rxi, dsr
	std::vector<Ort::Value> input_tensors = transform(aImg,
		nHeight,
		nWeigh,
		dynamic_input_node_dims,
		dynamic_src_value_handler,
		dynamic_r1i_value_handler,
		dynamic_r2i_value_handler,
		dynamic_r3i_value_handler,
		dynamic_r4i_value_handler,
		dynamic_dsr_value_handler
	);
	// ǰ������
	auto output_tensors = session->Run(Ort::RunOptions{ nullptr },
		input_node_names.data(),
		input_tensors.data(), num_inputs, output_node_names.data(),
		num_outputs
	);
	
	// ����Ľ������
	Ort::Value& fgr = output_tensors.at(0); // fgr (1,3,h,w) 0.~1.
	Ort::Value& pha = output_tensors.at(1); // pha (1,1,h,w) 0.~1.
	// �Զ���ȡά������
	auto fgr_dims = fgr.GetTypeInfo().GetTensorTypeAndShapeInfo().GetShape();
	auto pha_dims = pha.GetTypeInfo().GetTensorTypeAndShapeInfo().GetShape();
	const unsigned int height = fgr_dims.at(2); // output height
	const unsigned int width = fgr_dims.at(3); // output width
	const unsigned int target_tensor_size = height * width;
	
	unsigned char* imgResult = new unsigned char[height * width*3];
	
	
	
	// ��GPU��ִ�б����ڻ�����
	float* dstDevData;
	hipMalloc((void**)&dstDevData, sizeof(float) * target_tensor_size);
	hipMemcpy(dstDevData, pha.GetTensorData<float>(), sizeof(float) * target_tensor_size, hipMemcpyHostToDevice);
	unsigned char* srcDevData;
	hipMalloc((void**)&srcDevData, sizeof(unsigned char) * target_tensor_size*3);
	hipMemcpy(srcDevData, aImg, sizeof(unsigned char) * target_tensor_size*3, hipMemcpyHostToDevice);
	dim3 blockSize(32, 32);
    	dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
	// hwc to chw / bgr to rgb
	process<<<gridSize, blockSize>>>(srcDevData, dstDevData, height, width);
	hipDeviceSynchronize();  
	hipMemcpy(imgResult, srcDevData, sizeof(unsigned char) * target_tensor_size*3, hipMemcpyDeviceToHost);
	
	hipFree(srcDevData);
	hipFree(dstDevData);
	
	// ������������Ĵ�С
	size_t nimg_size = sizeof(unsigned char) * height * width*3;
	std::memcpy(aResultImg, imgResult, nimg_size);
	


	// ����ѭ�����������ġ���̬ά�ȸ���
	update_context(output_tensors,
		dynamic_input_node_dims,
		dynamic_src_value_handler,
		dynamic_r1i_value_handler,
		dynamic_r2i_value_handler,
		dynamic_r3i_value_handler,
		dynamic_r4i_value_handler,
		context_is_update
	);
}

#include "hip/hip_runtime.h"
#include "Create_tensor.cuh"

__global__ void process(const unsigned char* srcData, float* tgtData, const int h, const int w)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = ix + iy * w;
    int idx3 = idx * 3;

    if (ix < w && iy < h)
    {
        tgtData[idx] = static_cast<float>(srcData[idx3 + 2]) / 255.0f;
        tgtData[idx + h * w] = static_cast<float>(srcData[idx3 + 1]) / 255.0f;
        tgtData[idx + h * w * 2] = static_cast<float>(srcData[idx3]) / 255.0f;
    }
}

// ��������
Ort::Value create_tensor(unsigned char* aImg,
	const std::vector<int64_t>& tensor_dims,
	const Ort::MemoryInfo& memory_info_handler,
	std::vector<float>& tensor_value_handler)
	throw(std::runtime_error)
{
	const unsigned int target_channel = tensor_dims.at(1);
	const unsigned int target_height = tensor_dims.at(2);
	const unsigned int target_width = tensor_dims.at(3);
	const unsigned int target_tensor_size = target_channel * target_height * target_width;
	tensor_value_handler.resize(target_tensor_size);
	// ��GPU��ִ��toCHW����
	float* dstDevData;
	hipMalloc((void**)&dstDevData, sizeof(float) * target_tensor_size);
	unsigned char* srcDevData;
	hipMalloc((void**)&srcDevData, sizeof(unsigned char) * target_tensor_size);
	hipMemcpy(srcDevData, aImg, sizeof(unsigned char) * target_tensor_size, hipMemcpyHostToDevice);
	dim3 blockSize(32, 32);
    	dim3 gridSize((target_width + blockSize.x - 1) / blockSize.x, (target_height + blockSize.y - 1) / blockSize.y);
	// hwc to chw / bgr to rgb
	process<<<gridSize, blockSize>>>(srcDevData, dstDevData, target_height, target_width);
	hipDeviceSynchronize();  
	hipMemcpy(tensor_value_handler.data(), dstDevData, sizeof(float) * target_tensor_size, hipMemcpyDeviceToHost);
	
	hipFree(srcDevData);
	hipFree(dstDevData);

	return Ort::Value::CreateTensor<float>(memory_info_handler, tensor_value_handler.data(),
		target_tensor_size, tensor_dims.data(),
		tensor_dims.size());

}
